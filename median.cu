#include "hip/hip_runtime.h"
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <hip/hip_runtime.h>

#define WORKING_TYPE uchar
#define N_CHANNELS   1

constexpr int BLOCK_DIM_2D = 32;
constexpr int MEDIAN_WIN_SIZE = 5;
constexpr int MEDIAN_HALF_WIN_SIZE  = (MEDIAN_WIN_SIZE-1)/2;
constexpr int BLOCK_DIM_COMPUTE = BLOCK_DIM_2D-MEDIAN_HALF_WIN_SIZE;
constexpr int BLOCK_STEP = BLOCK_DIM_2D-2*MEDIAN_HALF_WIN_SIZE;

constexpr int MEDIAN_WIN_LEN = MEDIAN_WIN_SIZE*MEDIAN_WIN_SIZE;
__global__ void medianFilterKernel(const cv::cuda::PtrStepSz<WORKING_TYPE[N_CHANNELS]> input,
                                   cv::cuda::PtrStepSz<WORKING_TYPE[N_CHANNELS]> output,
                                   const int rows, const int cols)
{
    const int local_idx_y = threadIdx.y;
    const int local_idx_x = threadIdx.x;
    int row = blockIdx.y * BLOCK_STEP + local_idx_y - MEDIAN_HALF_WIN_SIZE;
    int col = blockIdx.x * BLOCK_STEP + local_idx_x - MEDIAN_HALF_WIN_SIZE;
    const int k = blockIdx.z * blockDim.z + threadIdx.z;
    if(k>=N_CHANNELS)
        return;
    __shared__ WORKING_TYPE sharedmem[BLOCK_DIM_2D][BLOCK_DIM_2D];  //initialize shared memory
    // take image values
    bool on_image = row>=0 && row<rows && col>=0 && col<cols;
    if(on_image)
        sharedmem[local_idx_y][local_idx_x] = input(row, col)[k];
     else
        sharedmem[local_idx_y][local_idx_x] = 0;
    __syncthreads();   // wait for all threads to be finished.

    if(!on_image)
        return;
    // check for borders
    if(local_idx_y<MEDIAN_HALF_WIN_SIZE || local_idx_x<MEDIAN_HALF_WIN_SIZE)
        return;
    if(local_idx_y>=BLOCK_DIM_COMPUTE || local_idx_x>=BLOCK_DIM_COMPUTE)
        return;
    // pick neighbors
    float vals[MEDIAN_WIN_LEN];
    for(int win_r = -MEDIAN_HALF_WIN_SIZE; win_r<=MEDIAN_HALF_WIN_SIZE; win_r++)
        for(int win_c = -MEDIAN_HALF_WIN_SIZE; win_c<=MEDIAN_HALF_WIN_SIZE; win_c++)
            vals[(MEDIAN_HALF_WIN_SIZE+win_r)*MEDIAN_WIN_SIZE+win_c+MEDIAN_HALF_WIN_SIZE] = sharedmem[local_idx_y+win_r][local_idx_x+win_c];

    // sorting
    for (int i = 0; i < MEDIAN_WIN_LEN; i++) {
        for (int j = i + 1; j < MEDIAN_WIN_LEN; j++) {
            if (vals[i] > vals[j]) {
                // swap
                float tmp = vals[i];
                vals[i] = vals[j];
                vals[j] = tmp;
            }
        }
    }

    output(row, col)[k] = vals[MEDIAN_WIN_LEN/2];   //Set the output image values.
}

extern
void medianFilterCu(const cv::cuda::GpuMat &src,
                    cv::cuda::GpuMat &dst,
                    cv::cuda::Stream &stream){

    hipStream_t c_stream = cv::cuda::StreamAccessor::getStream(stream);
    size_t n_layers = src.channels();
    assert(n_layers==N_CHANNELS);

    int rows = src.rows;
    int cols = src.cols;

    cv::cuda::createContinuous(src.size(), src.type(), dst);

    //take block and grids.
     dim3 dimBlock(BLOCK_DIM_2D, BLOCK_DIM_2D, 1);
     dim3 dimGrid((int)ceil(((float)cols+BLOCK_STEP) / (float)BLOCK_STEP),
                  (int)ceil(((float)rows+BLOCK_STEP) / (float)BLOCK_STEP),
                  static_cast<int>(std::ceil(N_CHANNELS / static_cast<double>(dimBlock.z))));

    medianFilterKernel<<<dimGrid, dimBlock, 0, c_stream>>>(src, dst, rows, cols);

}
